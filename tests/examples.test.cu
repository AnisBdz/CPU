#include <catch.hpp>
#include "cpu.hh"
#include "loader.hh"
#include "program.hh"
#include "instruction.hh"
#include "operand.hh"

TEST_CASE("example 01: mov, add, mov") {
    // mov al, [0x5]
    Instruction i1(instruction_code::mov, std::vector<operand_ptr> {
        std::make_shared<RegisterOperand>(register_code::al),
        std::make_shared<MemoryOperand>(5, 8)
    });


    // mov dl, [0x3]
    Instruction i2(instruction_code::mov, std::vector<operand_ptr> {
        std::make_shared<RegisterOperand>(register_code::dl),
        std::make_shared<MemoryOperand>(3, 8)
    });

    // add dl, al
    Instruction i3(instruction_code::add, std::vector<operand_ptr> {
        std::make_shared<RegisterOperand>(register_code::dl),
        std::make_shared<RegisterOperand>(register_code::al)
    });

    // add [0x8], al
    Instruction i4(instruction_code::mov, std::vector<operand_ptr> {
        std::make_shared<MemoryOperand>(8, 8),
        std::make_shared<RegisterOperand>(register_code::dl)
    });

    // add bh, [0x8]
    Instruction i5(instruction_code::mov, std::vector<operand_ptr> {
        std::make_shared<RegisterOperand>(register_code::bh),
        std::make_shared<MemoryOperand>(8, 8)
    });

    // hlt
    Instruction i6(instruction_code::hlt, std::vector<operand_ptr>{});

    Program program;
    program.add_instruction(i1);
    program.add_instruction(i2);
    program.add_instruction(i3);
    program.add_instruction(i4);
    program.add_instruction(i5);
    program.add_instruction(i6);
    program.add_data(std::vector<uint8_t>{0, 1, 2, 3, 4, 5, 6, 7, 1, 1, 1, 1, 1, 1, 1, 1, 1});


    RandomAccessMemory ram;
    CentralProcessingUnit cpu(ram);
    auto registers = cpu.get_registers();

    Loader loader(cpu, ram);
    loader.load(program);

    cpu.start();

    REQUIRE(registers[register_code::bh]->get_value() == 8);
}
